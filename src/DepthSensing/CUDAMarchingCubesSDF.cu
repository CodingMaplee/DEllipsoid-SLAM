#include "hip/hip_runtime.h"

#include <cutil_inline.h>
#include <cutil_math.h>


#include "MarchingCubesSDFUtil.h"


__global__ void resetMarchingCubesKernel(MarchingCubesData data) 
{
	*data.d_numTriangles = 0;
}

__global__ void extractIsoSurfaceKernel(HashDataStruct hashData, RayCastData rayCastData, MarchingCubesData data) 
{
	uint idx = blockIdx.x;

	const HashEntry& entry = hashData.d_hash[idx];
	if (entry.ptr != FREE_ENTRY) {
		int3 pi_base = hashData.SDFBlockToVirtualVoxelPos(entry.pos);
		int3 pi = pi_base + make_int3(threadIdx);
		float3 worldPos = hashData.virtualVoxelPosToWorld(pi);

		data.extractIsoSurfaceAtPosition(worldPos, hashData, rayCastData);
	}
}
__global__ void extractObjectSurfaceKernel(HashDataStruct hashData, RayCastData rayCastData, MarchingCubesData data, uchar objectCat)
{
	uint idx = blockIdx.x;

	const HashEntry& entry = hashData.d_hash[idx];

	if (entry.ptr != FREE_ENTRY) {
		int3 pi_base = hashData.SDFBlockToVirtualVoxelPos(entry.pos);
		int3 pi = pi_base + make_int3(threadIdx);
		float3 worldPos = hashData.virtualVoxelPosToWorld(pi);
		Voxel v = hashData.getVoxel(worldPos);
		if(v.catLabel == objectCat)
		{
			data.extractIsoSurfaceAtPosition(worldPos, hashData, rayCastData);
		}
	}
}
__global__ void clearObjectSurfaceKernel(HashDataStruct hashData, RayCastData rayCastData, MarchingCubesData data, uchar objectCat)
{
    uint idx = blockIdx.x;

    const HashEntry& entry = hashData.d_hash[idx];

    if (entry.ptr != FREE_ENTRY) {
        int3 pi_base = hashData.SDFBlockToVirtualVoxelPos(entry.pos);
        int3 pi = pi_base + make_int3(threadIdx);
        float3 worldPos = hashData.virtualVoxelPosToWorld(pi);
        Voxel v = hashData.getVoxel(worldPos);
        if(v.catLabel == objectCat)
        {
            int id = entry.ptr + hashData.virtualVoxelPosToLocalSDFBlockIndex(pi);
            hashData.deleteVoxel(id);
        }
    }
}
__global__ void extractDynamicSurfaceKernel(HashDataStruct hashData, RayCastData rayCastData, MarchingCubesData data)
{
    uint idx = blockIdx.x;

    const HashEntry& entry = hashData.d_hash[idx];
    if (entry.ptr != FREE_ENTRY) {
        int3 pi_base = hashData.SDFBlockToVirtualVoxelPos(entry.pos);
        int3 pi = pi_base + make_int3(threadIdx);
        float3 worldPos = hashData.virtualVoxelPosToWorld(pi);

        data.extractDynamicSurfaceAtPosition(worldPos, hashData, rayCastData);
    }
}


extern "C" void resetMarchingCubesCUDA(MarchingCubesData& data)
{
	const dim3 blockSize(1, 1, 1);
	const dim3 gridSize(1, 1, 1);

	resetMarchingCubesKernel<<<gridSize, blockSize>>>(data);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

extern "C" void extractIsoSurfaceCUDA(const HashDataStruct& hashData, const RayCastData& rayCastData, const MarchingCubesParams& params, MarchingCubesData& data)
{
	const dim3 gridSize(params.m_hashNumBuckets*params.m_hashBucketSize, 1, 1);
	const dim3 blockSize(params.m_sdfBlockSize, params.m_sdfBlockSize, params.m_sdfBlockSize);

	extractIsoSurfaceKernel<<<gridSize, blockSize>>>(hashData, rayCastData, data);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}
extern "C" void extractObjectSurfaceCUDA(const HashDataStruct& hashData, const RayCastData& rayCastData, const MarchingCubesParams& params, MarchingCubesData& data, uchar objCat)
{
	const dim3 gridSize(params.m_hashNumBuckets*params.m_hashBucketSize, 1, 1);
	const dim3 blockSize(params.m_sdfBlockSize, params.m_sdfBlockSize, params.m_sdfBlockSize);

	extractObjectSurfaceKernel<<<gridSize, blockSize>>>(hashData, rayCastData, data, objCat);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}
extern "C" void clearObjectSurfaceCUDA(const HashDataStruct& hashData, const RayCastData& rayCastData, const MarchingCubesParams& params, MarchingCubesData& data, uchar objCat)
{
    const dim3 gridSize(params.m_hashNumBuckets*params.m_hashBucketSize, 1, 1);
    const dim3 blockSize(params.m_sdfBlockSize, params.m_sdfBlockSize, params.m_sdfBlockSize);

    clearObjectSurfaceKernel<<<gridSize, blockSize>>>(hashData, rayCastData, data, objCat);

#ifdef _DEBUG
    cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}
extern "C" void extractDynamicSurfaceCUDA(const HashDataStruct& hashData, const RayCastData& rayCastData, const MarchingCubesParams& params, MarchingCubesData& data)
{
    const dim3 gridSize(params.m_hashNumBuckets*params.m_hashBucketSize, 1, 1);
    const dim3 blockSize(params.m_sdfBlockSize, params.m_sdfBlockSize, params.m_sdfBlockSize);

    extractDynamicSurfaceKernel<<<gridSize, blockSize>>>(hashData, rayCastData, data);

#ifdef _DEBUG
    cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}