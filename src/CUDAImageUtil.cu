#include "hip/hip_runtime.h"
#include "mLibCuda.h"
#include "CUDAImageUtil.h"


#define T_PER_BLOCK 16

#define MINF __int_as_float(0xff800000)


template<class T> void CUDAImageUtil::copy(T* d_output, T* d_input, unsigned int width, unsigned int height) {
	MLIB_CUDA_SAFE_CALL(hipMemcpy(d_output, d_input, sizeof(T)*width*height, hipMemcpyDeviceToDevice));
}
template<> void CUDAImageUtil::copy<float>(float* d_output, float* d_input, unsigned int width, unsigned int height) {
	MLIB_CUDA_SAFE_CALL(hipMemcpy(d_output, d_input, sizeof(float)*width*height, hipMemcpyDeviceToDevice));
}

template<> void CUDAImageUtil::copy<uchar4>(uchar4* d_output, uchar4* d_input, unsigned int width, unsigned int height) {
	MLIB_CUDA_SAFE_CALL(hipMemcpy(d_output, d_input, sizeof(uchar4)*width*height, hipMemcpyDeviceToDevice));
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Resample Float Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

inline __device__ float bilinearInterpolationFloat(float x, float y, const float* d_input, unsigned int imageWidth, unsigned int imageHeight)
{

	const int2 p00 = make_int2(floor(x), floor(y));
	const int2 p01 = p00 + make_int2(0.0f, 1.0f);
	const int2 p10 = p00 + make_int2(1.0f, 0.0f);
	const int2 p11 = p00 + make_int2(1.0f, 1.0f);

	const float alpha = x - p00.x;
	const float beta = y - p00.y;

	float s0 = 0.0f; float w0 = 0.0f;
	if (p00.x < imageWidth && p00.y < imageHeight) { float v00 = d_input[p00.y*imageWidth + p00.x]; if (v00 != MINF) { s0 += (1.0f - alpha)*v00; w0 += (1.0f - alpha); } }
	if (p10.x < imageWidth && p10.y < imageHeight) { float v10 = d_input[p10.y*imageWidth + p10.x]; if (v10 != MINF) { s0 += alpha *v10; w0 += alpha; } }

	float s1 = 0.0f; float w1 = 0.0f;
	if (p01.x < imageWidth && p01.y < imageHeight) { float v01 = d_input[p01.y*imageWidth + p01.x]; if (v01 != MINF) { s1 += (1.0f - alpha)*v01; w1 += (1.0f - alpha); } }
	if (p11.x < imageWidth && p11.y < imageHeight) { float v11 = d_input[p11.y*imageWidth + p11.x]; if (v11 != MINF) { s1 += alpha *v11; w1 += alpha; } }

	const float p0 = s0 / w0;
	const float p1 = s1 / w1;

	float ss = 0.0f; float ww = 0.0f;
	if (w0 > 0.0f) { ss += (1.0f - beta)*p0; ww += (1.0f - beta); }
	if (w1 > 0.0f) { ss += beta *p1; ww += beta; }

	if (ww > 0.0f) return ss / ww;
	else		  return MINF;
}

//template<class T>
//__global__ void resample_Kernel(T* d_output, T* d_input, unsigned int inputWidth, unsigned int inputHeight, unsigned int outputWidth, unsigned int outputHeight)
//{
//	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
//	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
//
//	if (x < outputWidth && y < outputHeight)
//	{
//		const float scaleWidth = (float)(inputWidth - 1) / (float)(outputWidth - 1);
//		const float scaleHeight = (float)(inputHeight - 1) / (float)(outputHeight - 1);
//
//		const unsigned int xInput = (unsigned int)(x*scaleWidth + 0.5f);
//		const unsigned int yInput = (unsigned int)(y*scaleHeight + 0.5f);
//
//		if (xInput < inputWidth && yInput < inputHeight)
//		{
//			if (std::is_same<T, float>::value) {
//				d_output[y*outputWidth + x] = (T)bilinearInterpolationFloat(x*scaleWidth, y*scaleHeight, (float*)d_input, inputWidth, inputHeight);
//			}
//			else if (std::is_same<T, uchar4>::value) {
//				d_output[y*outputWidth + x] = d_input[yInput*inputWidth + xInput];
//			}
//			else {
//				//static_assert(false, "bla");
//			}
//		}
//	}
//}
//
//template<class T> void CUDAImageUtil::resample(T* d_output, unsigned int outputWidth, unsigned int outputHeight, T* d_input, unsigned int inputWidth, unsigned int inputHeight) {
//
//	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1) / T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1) / T_PER_BLOCK);
//	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);
//
//	resample_Kernel << <gridSize, blockSize >> >(d_output, d_input, inputWidth, inputHeight, outputWidth, outputHeight);
//
//#ifdef _DEBUG
//	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
//	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
//#endif
//}


__global__ void resampleFloat_Kernel(float* d_output, unsigned int outputWidth, unsigned int outputHeight, const float* d_input, unsigned int inputWidth, unsigned int inputHeight)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < outputWidth && y < outputHeight)
	{
		const float scaleWidth = (float)(inputWidth-1) / (float)(outputWidth-1);
		const float scaleHeight = (float)(inputHeight-1) / (float)(outputHeight-1);

		const unsigned int xInput = (unsigned int)(x*scaleWidth + 0.5f);
		const unsigned int yInput = (unsigned int)(y*scaleHeight + 0.5f);

		if (xInput < inputWidth && yInput < inputHeight) {
			d_output[y*outputWidth + x] = d_input[yInput*inputWidth + xInput];
			//d_output[y*outputWidth + x] = bilinearInterpolationFloat(x*scaleWidth, y*scaleHeight, d_input, inputWidth, inputHeight);
		}
	}
}

void CUDAImageUtil::resampleFloat(float* d_output, unsigned int outputWidth, unsigned int outputHeight, const float* d_input, unsigned int inputWidth, unsigned int inputHeight) {

	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1) / T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	resampleFloat_Kernel << <gridSize, blockSize >> >(d_output, outputWidth, outputHeight, d_input, inputWidth, inputHeight);

#ifdef _DEBUG
	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#endif
}

__global__ void resampleFloat4_Kernel(float4* d_output, unsigned int outputWidth, unsigned int outputHeight, const float4* d_input, unsigned int inputWidth, unsigned int inputHeight)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < outputWidth && y < outputHeight)
	{
		const float scaleWidth = (float)(inputWidth-1) / (float)(outputWidth-1);
		const float scaleHeight = (float)(inputHeight-1) / (float)(outputHeight-1);

		const unsigned int xInput = (unsigned int)(x*scaleWidth + 0.5f);
		const unsigned int yInput = (unsigned int)(y*scaleHeight + 0.5f);

		if (xInput < inputWidth && yInput < inputHeight) {
			d_output[y*outputWidth + x] = d_input[yInput*inputWidth + xInput];
			//d_output[y*outputWidth + x] = bilinearInterpolationFloat(x*scaleWidth, y*scaleHeight, d_input, inputWidth, inputHeight);
		}
	}
}
void CUDAImageUtil::resampleFloat4(float4* d_output, unsigned int outputWidth, unsigned int outputHeight, const float4* d_input, unsigned int inputWidth, unsigned int inputHeight) {

	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1) / T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	resampleFloat4_Kernel << <gridSize, blockSize >> >(d_output, outputWidth, outputHeight, d_input, inputWidth, inputHeight);

#ifdef _DEBUG
	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#endif
}


__global__ void resampleUCHAR4_Kernel(uchar4* d_output, unsigned int outputWidth, unsigned int outputHeight, const uchar4* d_input, unsigned int inputWidth, unsigned int inputHeight)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < outputWidth && y < outputHeight)
	{
		const float scaleWidth = (float)(inputWidth-1) / (float)(outputWidth-1);
		const float scaleHeight = (float)(inputHeight-1) / (float)(outputHeight-1);

		const unsigned int xInput = (unsigned int)(x*scaleWidth + 0.5f);
		const unsigned int yInput = (unsigned int)(y*scaleHeight + 0.5f);

		if (xInput < inputWidth && yInput < inputHeight) {
			d_output[y*outputWidth + x] = d_input[yInput*inputWidth + xInput];
		}
	}
}

void CUDAImageUtil::resampleUCHAR4(uchar4* d_output, unsigned int outputWidth, unsigned int outputHeight, const uchar4* d_input, unsigned int inputWidth, unsigned int inputHeight) {

	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1) / T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	resampleUCHAR4_Kernel << <gridSize, blockSize >> >(d_output, outputWidth, outputHeight, d_input, inputWidth, inputHeight);

#ifdef _DEBUG
	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Color to Intensity
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__host__ __device__
float convertToIntensity(const uchar4& c) {
	return (0.299f*c.x + 0.587f*c.y + 0.114f*c.z) / 255.0f;
}



__global__ void convertUCHAR4ToIntensityFloat_Kernel(float* d_output, const uchar4* d_input, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		d_output[y*width + x] = convertToIntensity(d_input[y*width + x]);
	}
}

void CUDAImageUtil::convertUCHAR4ToIntensityFloat(float* d_output, const uchar4* d_input, unsigned int width, unsigned int height) {

	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	convertUCHAR4ToIntensityFloat_Kernel << <gridSize, blockSize >> >(d_output, d_input, width, height);

#ifdef _DEBUG
	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#endif
}

__global__ void resampleToIntensity_Kernel(float* d_output, unsigned int outputWidth, unsigned int outputHeight, const uchar4* d_input, unsigned int inputWidth, unsigned int inputHeight)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < outputWidth && y < outputHeight)
	{
		const float scaleWidth = (float)(inputWidth-1) / (float)(outputWidth-1);
		const float scaleHeight = (float)(inputHeight-1) / (float)(outputHeight-1);

		const unsigned int xInput = (unsigned int)(x*scaleWidth + 0.5f);
		const unsigned int yInput = (unsigned int)(y*scaleHeight + 0.5f);

		if (xInput < inputWidth && yInput < inputHeight) {
			d_output[y*outputWidth + x] = convertToIntensity(d_input[yInput*inputWidth + xInput]);
		}
	}
}

void CUDAImageUtil::resampleToIntensity(float* d_output, unsigned int outputWidth, unsigned int outputHeight, const uchar4* d_input, unsigned int inputWidth, unsigned int inputHeight) {

	const dim3 gridSize((outputWidth + T_PER_BLOCK - 1) / T_PER_BLOCK, (outputHeight + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	resampleToIntensity_Kernel << <gridSize, blockSize >> >(d_output, outputWidth, outputHeight, d_input, inputWidth, inputHeight);

#ifdef _DEBUG
	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// derivatives 
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void computeIntensityDerivatives_Kernel(float2* d_output, const float* d_input, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < width && y < height)
	{
		d_output[y*width + x] = make_float2(MINF, MINF);

		//derivative
		if (x > 0 && x < width - 1 && y > 0 && y < height - 1)
		{ 
			float pos00 = d_input[(y - 1)*width + (x - 1)]; if (pos00 == MINF) return;
			float pos01 = d_input[(y - 0)*width + (x - 1)];	if (pos01 == MINF) return;
			float pos02 = d_input[(y + 1)*width + (x - 1)];	if (pos02 == MINF) return;

			float pos10 = d_input[(y - 1)*width + (x - 0)]; if (pos10 == MINF) return;
			//float pos11 = d_input[(y-0)*width + (x-0)]; if (pos11 == MINF) return;
			float pos12 = d_input[(y + 1)*width + (x - 0)]; if (pos12 == MINF) return;

			float pos20 = d_input[(y - 1)*width + (x + 1)]; if (pos20 == MINF) return;
			float pos21 = d_input[(y - 0)*width + (x + 1)]; if (pos21 == MINF) return;
			float pos22 = d_input[(y + 1)*width + (x + 1)]; if (pos22 == MINF) return;

			float resU = (-1.0f)*pos00 + (1.0f)*pos20 +
				(-2.0f)*pos01 + (2.0f)*pos21 +
				(-1.0f)*pos02 + (1.0f)*pos22;
			resU /= 8.0f;

			float resV = (-1.0f)*pos00 + (-2.0f)*pos10 + (-1.0f)*pos20 +
				(1.0f)*pos02 + (2.0f)*pos12 + (1.0f)*pos22;
			resV /= 8.0f;

			d_output[y*width + x] = make_float2(resU, resV);
		}
	}
}

void CUDAImageUtil::computeIntensityDerivatives(float2* d_output, const float* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeIntensityDerivatives_Kernel << <gridSize, blockSize >> >(d_output, d_input, width, height);

#ifdef _DEBUG
	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#endif
}

__global__ void computeIntensityGradientMagnitude_Kernel(float* d_output, const float* d_input, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < width && y < height)
	{
		d_output[y*width + x] = MINF;

		//derivative
		if (x > 0 && x < width - 1 && y > 0 && y < height - 1)
		{ 
			float pos00 = d_input[(y - 1)*width + (x - 1)]; if (pos00 == MINF) return;
			float pos01 = d_input[(y - 0)*width + (x - 1)];	if (pos01 == MINF) return;
			float pos02 = d_input[(y + 1)*width + (x - 1)];	if (pos02 == MINF) return;

			float pos10 = d_input[(y - 1)*width + (x - 0)]; if (pos10 == MINF) return;
			//float pos11 = d_input[(y-0)*width + (x-0)]; if (pos11 == MINF) return;
			float pos12 = d_input[(y + 1)*width + (x - 0)]; if (pos12 == MINF) return;

			float pos20 = d_input[(y - 1)*width + (x + 1)]; if (pos20 == MINF) return;
			float pos21 = d_input[(y - 0)*width + (x + 1)]; if (pos21 == MINF) return;
			float pos22 = d_input[(y + 1)*width + (x + 1)]; if (pos22 == MINF) return;

			float resU = (-1.0f)*pos00 + (1.0f)*pos20 +
				(-2.0f)*pos01 + (2.0f)*pos21 +
				(-1.0f)*pos02 + (1.0f)*pos22;
			//resU /= 8.0f;

			float resV = (-1.0f)*pos00 + (-2.0f)*pos10 + (-1.0f)*pos20 +
				(1.0f)*pos02 + (2.0f)*pos12 + (1.0f)*pos22;
			//resV /= 8.0f;

			d_output[y*width + x] = sqrt(resU * resU + resV * resV);
		}
	}
}
void CUDAImageUtil::computeIntensityGradientMagnitude(float* d_output, const float* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeIntensityGradientMagnitude_Kernel << <gridSize, blockSize >> >(d_output, d_input, width, height);

#ifdef _DEBUG
	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#endif
}



////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Convert Depth to Camera Space Positions
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void convertDepthFloatToCameraSpaceFloat4_Kernel(float4* d_output, const float* d_input, Eigen::Matrix4f intrinsicsInv, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		d_output[y*width + x] = make_float4(MINF, MINF, MINF, MINF);

		float depth = d_input[y*width + x];

		if (depth != MINF)
		{
            Eigen::Vector4f trans = intrinsicsInv*Eigen::Vector4f((float)x*depth, (float)y*depth, depth, depth);
			float4 cameraSpace = make_float4(trans(0), trans(1), trans(2), trans(3));
			d_output[y*width + x] = make_float4(cameraSpace.x, cameraSpace.y, cameraSpace.w, 1.0f);
			//d_output[y*width + x] = make_float4(depthCameraData.kinectDepthToSkeleton(x, y, depth), 1.0f);
		}
	}
}

void CUDAImageUtil::convertDepthFloatToCameraSpaceFloat4(float4* d_output, const float* d_input, const Eigen::Matrix4f& intrinsicsInv, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	convertDepthFloatToCameraSpaceFloat4_Kernel << <gridSize, blockSize >> >(d_output, d_input, intrinsicsInv, width, height);

#ifdef _DEBUG
	MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
	MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#endif
}
// check ground
__global__ void genPlaneMask_Kernel(uint* planeMask, const float4* dots, float A, float B, float C, float D, uint width, uint height, Eigen::Matrix4f cam2world, float threshold)
{
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;
    float4 dot = dots[x+y*width];
    Eigen::Vector4f world_dot = cam2world * Eigen::Vector4f(dot.x, dot.y, dot.z, 1.0);
    if (dot.x==MINF ||dot.y == MINF || dot.z == MINF)
    {
        planeMask[x+y*width] = 0;
    }
    else
    {
        float distance = abs(world_dot.x()*A+world_dot.y()*B+world_dot.z()*C+D)/sqrt(A*A+B*B+C*C);
        if (distance < threshold)
            planeMask[x+y*width] = 1;
        else
            planeMask[x+y*width] = 0;
    }
}

void CUDAImageUtil::genPlaneMask(uint* planeMask, const float4* dots, float A, float B, float C, float D, uint width, uint height, Eigen::Matrix4f cam2world, float threshold)
{
    const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    genPlaneMask_Kernel <<<gridSize, blockSize >>>(planeMask, dots, A, B, C, D, width, height, cam2world, threshold);

    MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
    MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#ifdef _DEBUG
#endif
}
__global__ void genPersonMask_Kernel(uchar* g_currMaskMapGpu, uchar* personMask, uint width, uint height)
{
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;
    uchar catLabel = g_currMaskMapGpu[x+y*width];

    if (catLabel == PERSON_CAT )//||catLabel == 8//||catLabel == 5&&catLabel == 5 || catLabel == 8||catLabel == 8
    {
        personMask[x+y*width] = 255;
        //printf("catLabel: %d\n", catLabel);
    }
    else
    {
        personMask[x+y*width] = 0;
    }
}


void CUDAImageUtil::genPersonMask(uchar* g_currMaskMapGpu, uchar* personMask, uint width, uint height)
{
    const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    genPersonMask_Kernel <<<gridSize, blockSize >>>(g_currMaskMapGpu, personMask, width, height);

    MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
    MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#ifdef _DEBUG
#endif
}

__global__ void genObjectMask_Kernel(uchar* maskGPUAll, uchar* maskGPU, uint* maskPixelNumGPU , uint maskIndex, uint width, uint height)
{
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;
    uchar catLabel = maskGPUAll[x+y*width];

    if (catLabel == maskIndex )
    {
        maskGPU[x+y*width] = 255;
        uint old = atomicAdd(&maskPixelNumGPU[0], 1);
    }
    else
    {
        maskGPU[x+y*width] = 0;
    }
}


void CUDAImageUtil::genObjectMask(uchar* maskGPUAll, uchar* maskGPU, uint* maskPixelNumGPU , uint maskIndex, uint width, uint height)
{
    const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    genObjectMask_Kernel <<<gridSize, blockSize >>>(maskGPUAll, maskGPU, maskPixelNumGPU, maskIndex, width, height);

    MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
    MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#ifdef _DEBUG
#endif
}

__global__ void int2uint_Kernel(int* mask, uchar* output, uint width, uint height)
{
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;
    int catLabel = mask[x+y*width];
    //printf("catLabel: %d\n", catLabel);
    if (catLabel < 0)
    {
        output[x+y*width] = 0;
    }
    else
    {
        output[x+y*width] = catLabel;
    }
}
void CUDAImageUtil::int2uchar(int* mask, uchar* output, uint width, uint height)
{
    const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    int2uint_Kernel <<<gridSize, blockSize >>>(mask, output, width, height);

    MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
    MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#ifdef _DEBUG
#endif
}


__global__ void GenerateRgbKernel(uchar3* virtual_rgb,  const Eigen::Vector4f boxBbox, const HashDataStruct& hashData, const Eigen::Matrix4f camera_pose, const Eigen::Matrix4f intrinsics, uint width, uint height)
{

    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    if (x >= 640 || y >= 480) return;
        if (boxBbox.x()<x && x<boxBbox.z() && boxBbox.y()<y && y<boxBbox.w()) {
            const float oSet = c_hashParams.m_virtualVoxelSize;
            Eigen::Vector4f point;
            point.z() = 2;
            point.x() = (x - intrinsics(0, 2)) * point.z() / intrinsics(0, 0);
            point.y() = (y - intrinsics(1, 2)) * point.z() / intrinsics(1, 1);
            point.w() = 1.0;
            point = camera_pose * point;
            float3 pos;
            pos.x = point.x();
            pos.y = point.y();
            pos.z = point.z();
           // const float3 posDual = pos-make_float3(oSet/2.0f, oSet/2.0f, oSet/2.0f);
            //Voxel v = hashData.getVoxel(posDual);
//            //printf("x=%d y=%d\n",x,y);
//            float current_depth = 1;
//            Eigen::Vector4f point;

            printf ("point x= %f y= %f z= %f\n", point.x(), point.y(), point.z());
             //float3 pos;
//            pos.x = point.x();
//            pos.y = point.y();
//            pos.z = point.z();
//            if (hashData.voxelExists(pos)){
//                //const HashEntry& entry = hashData.getHashEntry(pos);
            }

            //const HashEntry& entry = hashData.getHashEntry(pos);
//            printf("entry: %d %d %d\n", entry.pos.x, entry.pos.y, entry.pos.z);
//            int3 pi = hashData.SDFBlockToVirtualVoxelPos(entry.pos);
//            printf("pos: %d %d %d\n",pi.x,pi.y,pi.z);
//            Voxel voxel = hashData.getVoxel(pi);
}
void CUDAImageUtil:: GenerateRgb(uchar3* virtual_rgb, const Eigen::Vector4f boxBbox, const HashDataStruct& hashData, const Eigen::Matrix4f& camera_pose, const Eigen::Matrix4f& intrinsics ,uint width, uint height)
{
    const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);
    GenerateRgbKernel << <gridSize, blockSize >> >(virtual_rgb, boxBbox, hashData, camera_pose, intrinsics, width, height);
    #ifdef _DEBUG
    cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
    #endif
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Compute Normal Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void computeNormals_Kernel(float4* d_output, const float4* d_input, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	d_output[y*width + x] = make_float4(MINF, MINF, MINF, MINF);

	if (x > 0 && x < width - 1 && y > 0 && y < height - 1)
	{
		const float4 CC = d_input[(y + 0)*width + (x + 0)];
		const float4 PC = d_input[(y + 1)*width + (x + 0)];
		const float4 CP = d_input[(y + 0)*width + (x + 1)];
		const float4 MC = d_input[(y - 1)*width + (x + 0)];
		const float4 CM = d_input[(y + 0)*width + (x - 1)];

		if (CC.x != MINF && PC.x != MINF && CP.x != MINF && MC.x != MINF && CM.x != MINF)
		{
			const float3 n = cross(make_float3(PC) - make_float3(MC), make_float3(CP) - make_float3(CM));
			const float  l = length(n);

			if (l > 0.0f)
			{
				d_output[y*width + x] = make_float4(n / -l, 0.0f);
			}
		}
	}
}

void CUDAImageUtil::computeNormals(float4* d_output, const float4* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeNormals_Kernel << <gridSize, blockSize >> >(d_output, d_input, width, height);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

__global__ void computeNormalsSobel_Kernel(float4* d_output, const float4* d_input, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	d_output[y*width + x] = make_float4(MINF, MINF, MINF, MINF);

	if (x > 0 && x < width - 1 && y > 0 && y < height - 1)
	{
		float4 pos00 = d_input[(y - 1)*width + (x - 1)]; if (pos00.x == MINF) return;
		float4 pos01 = d_input[(y - 0)*width + (x - 1)]; if (pos01.x == MINF) return;
		float4 pos02 = d_input[(y + 1)*width + (x - 1)]; if (pos02.x == MINF) return;

		float4 pos10 = d_input[(y - 1)*width + (x - 0)]; if (pos10.x == MINF) return;
		//float4 pos11 = d_input[(y-0)*width + (x-0)]; if (pos11.x == MINF) return;
		float4 pos12 = d_input[(y + 1)*width + (x - 0)]; if (pos12.x == MINF) return;

		float4 pos20 = d_input[(y - 1)*width + (x + 1)]; if (pos20.x == MINF) return;
		float4 pos21 = d_input[(y - 0)*width + (x + 1)]; if (pos21.x == MINF) return;
		float4 pos22 = d_input[(y + 1)*width + (x + 1)]; if (pos22.x == MINF) return;

		float4 resU = (-1.0f)*pos00 + (1.0f)*pos20 +
			(-2.0f)*pos01 + (2.0f)*pos21 +
			(-1.0f)*pos02 + (1.0f)*pos22;

		float4 resV = (-1.0f)*pos00 + (-2.0f)*pos10 + (-1.0f)*pos20 +
			(1.0f)*pos02 + (2.0f)*pos12 + (1.0f)*pos22;

		const float3 n = cross(make_float3(resU.x, resU.y, resU.z), make_float3(resV.x, resV.y, resV.z));
		const float  l = length(n);

		if (l > 0.0f) d_output[y*width + x] = make_float4(n / l, 0.0f);
	}
}

void CUDAImageUtil::computeNormalsSobel(float4* d_output, const float4* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	computeNormalsSobel_Kernel << <gridSize, blockSize >> >(d_output, d_input, width, height);

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

__global__ void convertNormalsFloat4ToUCHAR4_Kernel(uchar4* d_output, const float4* d_input, unsigned int width, unsigned int height)
{
	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		d_output[y*width + x] = make_uchar4(0, 0, 0, 0);

		float4 p = d_input[y*width + x];

		if (p.x != MINF)
		{
			p = (p + 1.0f) / 2.0f; // -> [0, 1]
			d_output[y*width + x] = make_uchar4((uchar)round(p.x * 255), (uchar)round(p.y * 255), (uchar)round(p.z * 255), 0);
		}
	}
}

void CUDAImageUtil::convertNormalsFloat4ToUCHAR4(uchar4* d_output, const float4* d_input, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	convertNormalsFloat4ToUCHAR4_Kernel << <gridSize, blockSize >> >(d_output, d_input, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Joint Bilateral Filter
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

inline __device__ float gaussD(float sigma, int x, int y)
{
	return exp(-((x*x + y*y) / (2.0f*sigma*sigma)));
}
inline __device__ float gaussR(float sigma, float dist)
{
	return exp(-(dist*dist) / (2.0*sigma*sigma));
}

__global__ void bilateralFilterUCHAR4_Kernel(uchar4* d_output, uchar4* d_color, float* d_depth, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	const int kernelRadius = (int)ceil(2.0*sigmaD);

	d_output[y*width + x] = d_color[y*width + x];

	float3 sum = make_float3(0.0f, 0.0f, 0.0f);
	float sumWeight = 0.0f;

	const float depthCenter = d_depth[y*width + x];
	if (depthCenter != MINF)
	{
		for (int m = x - kernelRadius; m <= x + kernelRadius; m++)
		{
			for (int n = y - kernelRadius; n <= y + kernelRadius; n++)
			{
				if (m >= 0 && n >= 0 && m < width && n < height)
				{
					const uchar4 cur = d_color[n*width + m];
					const float currentDepth = d_depth[n*width + m];

					if (currentDepth != MINF) {
						const float weight = gaussD(sigmaD, m - x, n - y)*gaussR(sigmaR, currentDepth - depthCenter);

						sumWeight += weight;
						sum += weight*make_float3(cur.x, cur.y, cur.z);
					}
				}
			}
		}

		if (sumWeight > 0.0f) {
			float3 res = sum / sumWeight;
			d_output[y*width + x] = make_uchar4((uchar)res.x, (uchar)res.y, (uchar)res.z, 255);
		}
	}
}

void CUDAImageUtil::jointBilateralFilterColorUCHAR4(uchar4* d_output, uchar4* d_input, float* d_depth, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	bilateralFilterUCHAR4_Kernel << <gridSize, blockSize >> >(d_output, d_input, d_depth, sigmaD, sigmaR, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

__global__ void bilateralFilterFloat_Kernel(float* d_output, float* d_input, float* d_depth, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	const int kernelRadius = (int)ceil(2.0*sigmaD);

	d_output[y*width + x] = MINF;

	float sum = 0.0f;
	float sumWeight = 0.0f;

	const float depthCenter = d_depth[y*width + x];
	if (depthCenter != MINF)
	{
		for (int m = x - kernelRadius; m <= x + kernelRadius; m++)
		{
			for (int n = y - kernelRadius; n <= y + kernelRadius; n++)
			{
				if (m >= 0 && n >= 0 && m < width && n < height)
				{
					const float cur = d_input[n*width + m];
					const float currentDepth = d_depth[n*width + m];

					if (currentDepth != MINF && fabs(depthCenter - currentDepth) < sigmaR)
					{ //const float weight = gaussD(sigmaD, m - x, n - y)*gaussR(sigmaR, currentDepth - depthCenter);
						const float weight = gaussD(sigmaD, m - x, n - y);
						sumWeight += weight;
						sum += weight*cur;
					}
				}
			}
		}

		if (sumWeight > 0.0f) d_output[y*width + x] = sum / sumWeight;
	}
}
void CUDAImageUtil::jointBilateralFilterFloat(float* d_output, float* d_input, float* d_depth, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	bilateralFilterFloat_Kernel << <gridSize, blockSize >> >(d_output, d_input, d_depth, sigmaD, sigmaR, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

__global__ void adaptiveBilateralFilterIntensity_Kernel(float* d_output, const float* d_input, const float* d_depth, float sigmaD, float sigmaR, float adaptFactor, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	d_output[y*width + x] = MINF;

	float sum = 0.0f;
	float sumWeight = 0.0f;

	const float depthCenter = d_depth[y*width + x];
	if (depthCenter != MINF)
	{
		const float curSigma = sigmaD * adaptFactor / depthCenter;
		const int kernelRadius = (int)ceil(2.0*curSigma);

		for (int m = x - kernelRadius; m <= x + kernelRadius; m++)
		{
			for (int n = y - kernelRadius; n <= y + kernelRadius; n++)
			{
				if (m >= 0 && n >= 0 && m < width && n < height)
				{
					const float cur = d_input[n*width + m];
					const float currentDepth = d_depth[n*width + m];

					if (currentDepth != MINF && fabs(depthCenter - currentDepth) < sigmaR)
					{ //const float weight = gaussD(curSigma, m - x, n - y)*gaussR(sigmaR, currentDepth - depthCenter);
						const float weight = gaussD(curSigma, m - x, n - y);
						sumWeight += weight;
						sum += weight*cur;
					}
				}
			}
		}

		if (sumWeight > 0.0f) d_output[y*width + x] = sum / sumWeight;
	}
}
void CUDAImageUtil::adaptiveBilateralFilterIntensity(float* d_output, const float* d_input, const float* d_depth, float sigmaD, float sigmaR, float adaptFactor, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	adaptiveBilateralFilterIntensity_Kernel << <gridSize, blockSize >> >(d_output, d_input, d_depth, sigmaD, sigmaR, adaptFactor, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Erode Depth Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void erodeDepthMapDevice(float* d_output, float* d_input, int structureSize, int width, int height, float dThresh, float fracReq)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;


	if (x >= 0 && x < width && y >= 0 && y < height)
	{


		unsigned int count = 0;

		float oldDepth = d_input[y*width + x];
		for (int i = -structureSize; i <= structureSize; i++)
		{
			for (int j = -structureSize; j <= structureSize; j++)
			{
				if (x + j >= 0 && x + j < width && y + i >= 0 && y + i < height)
				{
					float depth = d_input[(y + i)*width + (x + j)];
					if (depth == MINF || depth == 0.0f || fabs(depth - oldDepth) > dThresh)
					{
						count++;
						//d_output[y*width+x] = MINF;
						//return;
					}
				}
			}
		}

		unsigned int sum = (2 * structureSize + 1)*(2 * structureSize + 1);
		if ((float)count / (float)sum >= fracReq) {
			d_output[y*width + x] = MINF;
		}
		else {
			d_output[y*width + x] = d_input[y*width + x];
		}
	}
}

void CUDAImageUtil::erodeDepthMap(float* d_output, float* d_input, int structureSize, unsigned int width, unsigned int height, float dThresh, float fracReq)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	erodeDepthMapDevice << <gridSize, blockSize >> >(d_output, d_input, structureSize, width, height, dThresh, fracReq);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}



////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Gauss Filter Float Map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void gaussFilterDepthMapDevice(float* d_output, const float* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	const int kernelRadius = (int)ceil(2.0*sigmaD);

	d_output[y*width + x] = MINF;

	float sum = 0.0f;
	float sumWeight = 0.0f;

	const float depthCenter = d_input[y*width + x];
	if (depthCenter != MINF)
	{
		for (int m = x - kernelRadius; m <= x + kernelRadius; m++)
		{
			for (int n = y - kernelRadius; n <= y + kernelRadius; n++)
			{
				if (m >= 0 && n >= 0 && m < width && n < height)
				{
					const float currentDepth = d_input[n*width + m];

					if (currentDepth != MINF && fabs(depthCenter - currentDepth) < sigmaR)
					{
						const float weight = gaussD(sigmaD, m - x, n - y);

						sumWeight += weight;
						sum += weight*currentDepth;
					}
				}
			}
		}
	}

	if (sumWeight > 0.0f) d_output[y*width + x] = sum / sumWeight;
}

void CUDAImageUtil::gaussFilterDepthMap(float* d_output, const float* d_input, float sigmaD, float sigmaR, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	gaussFilterDepthMapDevice << <gridSize, blockSize >> >(d_output, d_input, sigmaD, sigmaR, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

__global__ void gaussFilterIntensityDevice(float* d_output, const float* d_input, float sigmaD, unsigned int width, unsigned int height)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	const int kernelRadius = (int)ceil(2.0*sigmaD);

	//d_output[y*width + x] = MINF;

	float sum = 0.0f;
	float sumWeight = 0.0f;

	//const float center = d_input[y*width + x];
	//if (center != MINF) {
	for (int m = x - kernelRadius; m <= x + kernelRadius; m++)
	{
		for (int n = y - kernelRadius; n <= y + kernelRadius; n++)
		{
			if (m >= 0 && n >= 0 && m < width && n < height)
			{
				const float current = d_input[n*width + m];

				//if (current != MINF && fabs(center - current) < sigmaR) {
				const float weight = gaussD(sigmaD, m - x, n - y);

				sumWeight += weight;
				sum += weight*current;
				//}
			}
		}
	}
	//}

	if (sumWeight > 0.0f) d_output[y*width + x] = sum / sumWeight;
}

void CUDAImageUtil::gaussFilterIntensity(float* d_output, const float* d_input, float sigmaD, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	gaussFilterIntensityDevice << <gridSize, blockSize >> >(d_output, d_input, sigmaD, width, height);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// adaptive gauss filter float map
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void adaptiveGaussFilterDepthMap_Kernel(float* d_output, const float* d_input, float sigmaD, float sigmaR,
	unsigned int width, unsigned int height, float adaptFactor)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;


	d_output[y*width + x] = MINF;

	float sum = 0.0f;
	float sumWeight = 0.0f;

	const float depthCenter = d_input[y*width + x];
	if (depthCenter != MINF)
	{
		const float curSigma = sigmaD / depthCenter * adaptFactor;
		const int kernelRadius = (int)ceil(2.0*curSigma);

		for (int m = x - kernelRadius; m <= x + kernelRadius; m++)
		{
			for (int n = y - kernelRadius; n <= y + kernelRadius; n++)
			{
				if (m >= 0 && n >= 0 && m < width && n < height)
				{
					const float currentDepth = d_input[n*width + m];

					if (currentDepth != MINF && fabs(depthCenter - currentDepth) < sigmaR)
					{
						const float weight = gaussD(curSigma, m - x, n - y);

						sumWeight += weight;
						sum += weight*currentDepth;
					}
				}
			}
		}
	}

	if (sumWeight > 0.0f) d_output[y*width + x] = sum / sumWeight;
}
void CUDAImageUtil::adaptiveGaussFilterDepthMap(float* d_output, const float* d_input, float sigmaD, float sigmaR, float adaptFactor, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	adaptiveGaussFilterDepthMap_Kernel << <gridSize, blockSize >> >(d_output, d_input, sigmaD, sigmaR, width, height, adaptFactor);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

__global__ void adaptiveGaussFilterIntensity_Kernel(float* d_output, const float* d_input, const float* d_depth, float sigmaD,
	unsigned int width, unsigned int height, float adaptFactor)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	float sum = 0.0f;
	float sumWeight = 0.0f;

	d_output[y*width + x] = MINF; //(should not be used in the case of no valid depth)

	const float depthCenter = d_depth[y*width + x];
	if (depthCenter != MINF)
	{
		const float curSigma = sigmaD / depthCenter * adaptFactor;
		const int kernelRadius = (int)ceil(2.0*curSigma);

		for (int m = x - kernelRadius; m <= x + kernelRadius; m++)
		{
			for (int n = y - kernelRadius; n <= y + kernelRadius; n++)
			{
				if (m >= 0 && n >= 0 && m < width && n < height)
				{
					const float currentDepth = d_depth[n*width + m];
					if (currentDepth != MINF) // && fabs(depthCenter - currentDepth) < sigmaR)
					{
						const float current = d_input[n*width + m];
						const float weight = gaussD(curSigma, m - x, n - y);

						sumWeight += weight;
						sum += weight*current;
					}
				}
			}
		}
	}

	if (sumWeight > 0.0f) d_output[y*width + x] = sum / sumWeight;
}
void CUDAImageUtil::adaptiveGaussFilterIntensity(float* d_output, const float* d_input, const float* d_depth, float sigmaD, float adaptFactor, unsigned int width, unsigned int height)
{
	const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	adaptiveGaussFilterIntensity_Kernel << <gridSize, blockSize >> >(d_output, d_input, d_depth, sigmaD, width, height, adaptFactor);
#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

/////////////////////////////////////////////////////////////////////dynamic reconstruction

__global__ void extractMotionConsistency_Kernel(float* consistency_px, uchar* output_px, float threshold, const uchar* existingDynamicPx,
                                                const float* d_inputTar, const float* d_inputSrc,
                                                const float* d_intensityTar, const float* d_intensitySrc,
                                                const float* d_raftU, const float* d_raftV,
                                                Eigen::Matrix4f intrinsics, Eigen::Matrix4f transformLast, Eigen::Matrix4f transformCurrent_inv,
                                                unsigned int width, unsigned int height)
{
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x < width && y < height) {

        float depthSrc = d_inputSrc[y * width + x];
        //float depthOF = bilinearInterpolationFloat(x - d_raftU[y * width + x], y - d_raftV[y * width + x], d_inputTar, width, height);
        if(depthSrc == MINF || existingDynamicPx[y * width + x] == 255)//   || depthOF == MINF
        {
            consistency_px[y*width + x] = 0;
            output_px[y*width + x] = 0;
            return;
        }

        float4 dotSrc = make_float4(depthSrc * (x - intrinsics(0,2)) / intrinsics(0,0),
                                    depthSrc * (y - intrinsics(1,2)) / intrinsics(1,1), depthSrc, 1.0);
        //next, calculate the transformed 3D point in current camera pose.
        Eigen::Vector4f trans_cur = transformCurrent_inv * transformLast * Eigen::Vector4f(dotSrc.x, dotSrc.y, dotSrc.z, 1.0);
        float4 dotSrc2World2Current = make_float4(trans_cur(0), trans_cur(1), trans_cur(2), trans_cur(3));

        //project the transformed point the 2D.
        float2 screenPos = make_float2(
                dotSrc2World2Current.x*intrinsics(0,0)/dotSrc2World2Current.z + intrinsics(0,2),
                dotSrc2World2Current.y*intrinsics(1,1)/dotSrc2World2Current.z + intrinsics(1,2));
        //float4 projectDot = intrinsics * dotTrans / dotTrans.z;

        if(screenPos.x < width && screenPos.x > 0 && screenPos.y < height && screenPos.y > 0) {
            //printf("screenPos.x: %f, x: %d, screenPos.y: %f, y: %d\n", screenPos.x, x, screenPos.y, y);
            //calculate the consistency of optical flow.
            float optical_flow_consistency_x = screenPos.x - ((float) x - d_raftU[y * width + x]);
            float optical_flow_consistency_y = screenPos.y - ((float) y - d_raftV[y * width + x]);
            //printf("optical_flow_consistency_x:%f, optical_flow_consistency_y:%f\n", optical_flow_consistency_x, optical_flow_consistency_y);
            float photoSrc = d_intensitySrc[y * width + x] / 255;
            float photoTar1 = bilinearInterpolationFloat(screenPos.x, screenPos.y, d_intensityTar, width, height) / 255;
            float photoTar2 = bilinearInterpolationFloat(x - d_raftU[y * width + x],
                                                         y - d_raftV[y * width + x], d_intensityTar, width,
                                                         height) / 255;
            float intensity_consistency = sqrt((photoTar1 - photoSrc) * (photoTar1 - photoSrc));

            //printf("projectDot x:%f, projectDot y:%f, x:%d, y:%d \n", projectDot.x, projectDot.y, x, y);
            //printf("consistency:%f\n", sqrt(optical_flow_consistency_x*optical_flow_consistency_x+optical_flow_consistency_y*optical_flow_consistency_y));//+ sqrt(intensity_consistency * intensity_consistency)
            float consistency = sqrt(optical_flow_consistency_x * optical_flow_consistency_x +
                                     optical_flow_consistency_y * optical_flow_consistency_y);//+intensity_consistency

            //printf("consistency: %f, intensity: %f\n", consistency, intensity_consistency);
            consistency_px[y * width + x] = consistency;
            if(consistency > threshold && consistency < 9999)
            {
                output_px[y * width + x] = 255;
                //printf("screenPos.x: %f, x: %d, screenPos.y: %f, y: %d, raftu: %f, raftv: %f, consistency: %f\n", screenPos.x, x, screenPos.y, y, d_raftU[y * width + x], d_raftV[y * width + x], consistency);

            }
            //printf("screenPos.x: %f, x: %d, screenPos.y: %f, y: %d, raftu: %f, raftv: %f, consistency: %f\n", screenPos.x, x, screenPos.y, y, d_raftU[y * width + x], d_raftV[y * width + x], consistency);

        }
    }
}

void CUDAImageUtil::extractMotionConsistency(float* consistency_px, uchar* d_output, float threshold, const uchar* existingDynamicPx, const float* d_inputTar, const float* d_inputSrc, const float* d_intensityTar, const float* d_intensitySrc,
                                               const float* d_raftU, const float* d_raftV,const Eigen::Matrix4f& intrinsics,
                                               const Eigen::Matrix4f& transformLast, const Eigen::Matrix4f& transformCurrent_inv,
                                               unsigned int width, unsigned int height)
{
    const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    extractMotionConsistency_Kernel <<<gridSize, blockSize >>>(consistency_px, d_output, threshold, existingDynamicPx, d_inputTar, d_inputSrc, d_intensityTar, d_intensitySrc,
                                                               d_raftU, d_raftV, intrinsics, transformLast, transformCurrent_inv,
                                                               width, height);
    MLIB_CUDA_SAFE_CALL(hipDeviceSynchronize());
    MLIB_CUDA_CHECK_ERR(__FUNCTION__);
#ifdef _DEBUG

#endif
}
/*
void CUDAImageUtil::checkNeighbours(float* d_output, const float* d_input, const float* d_depth, float sigmaD, float adaptFactor, unsigned int width, unsigned int height)
{
    const dim3 gridSize((width + T_PER_BLOCK - 1) / T_PER_BLOCK, (height + T_PER_BLOCK - 1) / T_PER_BLOCK);
    const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

    adaptiveGaussFilterIntensity_Kernel << <gridSize, blockSize >> >(d_output, d_input, d_depth, sigmaD, width, height, adaptFactor);
#ifdef _DEBUG
    cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}*/
